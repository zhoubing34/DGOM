#include "hip/hip_runtime.h"
/* -*- mode: C; c-basic-offset: 8; c-indent-level: 8; c-continued-statement-offset: 8; c-label-offset: -8; -*- */

#include <stdio.h>
#include <hip/hip_runtime.h>

texture<float4, 1, hipReadModeElementType> t_LIFT;
texture<float4, 1, hipReadModeElementType> t_DrDs;
texture<float, 1, hipReadModeElementType> t_vgeo;
texture<float, 1, hipReadModeElementType> t_Q;
texture<float, 1, hipReadModeElementType> t_partQ;
texture<float, 1, hipReadModeElementType> t_surfinfo;

static float *c_LIFT;
static float4 *c_DrDs;
static float2 *c_DrDs2;
static float *c_surfinfo;
static float *c_vgeo;
static float *c_Q; 
static float *c_partQ; 
static float *c_rhsQ; 
static float *c_resQ; 
static float *c_tmp;

extern "C"
{

#include "fem.h"

double InitGPU2d(Mesh *mesh, int Nfields){

  printf("Np = %d, BSIZE = %d\n", p_Np, BSIZE);

  /* Q  */
  int sz = mesh->K*(BSIZE)*Nfields*sizeof(float); 

  float *f_Q = (float*) calloc(mesh->K*BSIZE*Nfields, sizeof(float));
  hipMalloc  ((void**) &c_Q, sz);
  hipMalloc  ((void**) &c_rhsQ, sz);
  hipMalloc  ((void**) &c_resQ, sz);
  hipMalloc  ((void**) &c_tmp, sz);
  hipMemcpy( c_Q,    f_Q, sz, hipMemcpyHostToDevice);
  hipMemcpy( c_rhsQ, f_Q, sz, hipMemcpyHostToDevice);
  hipMemcpy( c_resQ, f_Q, sz, hipMemcpyHostToDevice);
  hipMemcpy( c_tmp,  f_Q, sz, hipMemcpyHostToDevice);
  
  hipBindTexture(0,  t_Q, c_Q, sz); 
  
  sz = mesh->parNtotalout*sizeof(float);
  hipMalloc((void**) &c_partQ, sz);
  hipBindTexture(0,  t_partQ, c_partQ, sz); 

  /*  LIFT  */
  sz = p_Np*(p_Nfp)*(p_Nfaces+1)*sizeof(float);
#if 0
   float *f_LIFT = (float*) malloc(sz);
   int skL = 0;

   for(int m=0;m<p_Nfp*p_Nfaces;++m){
     for(int n=0;n<p_Np;++n){
       f_LIFT[skL++] = mesh->LIFT[n][m];
     }
   }
#else
   float *f_LIFT = (float*) malloc(sz);
   int skL = 0;
   for(int m=0;m<p_Nfp;++m){
     for(int n=0;n<p_Np;++n){
       for(int f=0;f<p_Nfaces;++f){
	 f_LIFT[skL++] = mesh->LIFT[0][p_Nfp*p_Nfaces*n+(f+p_Nfaces*m)];
       }
       ++skL;
     }
   }
#endif
   hipMalloc  ((void**) &c_LIFT, sz);
   hipMemcpy( c_LIFT, f_LIFT, sz, hipMemcpyHostToDevice);

#if 1
   /* Bind the array to the texture */
   hipBindTexture(0,  t_LIFT, c_LIFT, sz);

   /* DrDsDt */
   sz = BSIZE*BSIZE*4*sizeof(float);

   float* h_DrDs = (float*) calloc(BSIZE*BSIZE, sizeof(float4));
   int sk = 0;
   /* note transposed arrays to avoid "bank conflicts" */
   for(int n=0;n<p_Np;++n){ 
     for(int m=0;m<p_Np;++m){
       h_DrDs[4*(m+n*BSIZE)+0] = mesh->Dr[0][n+m*p_Np];
       h_DrDs[4*(m+n*BSIZE)+1] = mesh->Ds[0][n+m*p_Np];
#if (p_Np%2)==0
       h_DrDs[4*(m+n*BSIZE)+2] = mesh->Dr[0][n+1+m*p_Np];
       h_DrDs[4*(m+n*BSIZE)+3] = mesh->Ds[0][n+1+m*p_Np];
#endif

     }
   }
	   
   hipMalloc  ((void**) &c_DrDs, sz);
   hipMemcpy( c_DrDs, h_DrDs, sz, hipMemcpyHostToDevice);

   /* Bind the array to the texture */
   hipBindTexture(0,  t_DrDs, c_DrDs, sz);

   sz = BSIZE*BSIZE*2*sizeof(float);
   float* h_DrDs2 = (float*) calloc(BSIZE*BSIZE, sizeof(float2));
   sk = 0;
   /* note transposed arrays to avoid "bank conflicts" */
   for(int n=0;n<p_Np;++n){ 
     for(int m=0;m<p_Np;++m){
       h_DrDs2[2*(m+n*BSIZE)+0] = mesh->Dr[0][n+m*p_Np]; 
       h_DrDs2[2*(m+n*BSIZE)+1] = mesh->Ds[0][n+m*p_Np]; 
     }
   }
   hipMalloc  ((void**) &c_DrDs2, sz);
   hipMemcpy( c_DrDs2, h_DrDs2, sz, hipMemcpyHostToDevice);
   
   free(h_DrDs);

   /* vgeo */
   double drdx, dsdx, drdy, dsdy, J;
   float *vgeo = (float*) calloc(4*mesh->K, sizeof(float));

   for(int k=0;k<mesh->K;++k){
     GeometricFactors2d(mesh, k, &drdx, &dsdx, &drdy, &dsdy, &J);
     vgeo[k*4+0] = drdx;
     vgeo[k*4+1] = drdy;
     vgeo[k*4+2] = dsdx;
     vgeo[k*4+3] = dsdy;
   }

   sz = mesh->K*4*sizeof(float);
   hipMalloc  ((void**) &c_vgeo, sz);
   hipMemcpy( c_vgeo, vgeo, sz, hipMemcpyHostToDevice);
   hipBindTexture(0,  t_vgeo, c_vgeo, sz);
   
   /* surfinfo (vmapM, vmapP, Fscale, Bscale, nx, ny, nz, 0) */
   sz = mesh->K*p_Nfp*p_Nfaces*6*sizeof(float); 
   float* h_surfinfo = (float*) malloc(sz); 
   
   /* local-local info */
   sk = 0;
   int skP = -1;
   double *nxk = BuildVector(mesh->Nfaces);
   double *nyk = BuildVector(mesh->Nfaces);
   double *sJk = BuildVector(mesh->Nfaces);

   double dt = 1e6;

   for(int k=0;k<mesh->K;++k){
     GeometricFactors2d(mesh, k, &drdx, &dsdx, &drdy, &dsdy, &J);     
     Normals2d(mesh, k, nxk, nyk, sJk);
     
     for(int f=0;f<mesh->Nfaces;++f){
       dt = min(dt, J/sJk[f]);
  
       for(int m=0;m<p_Nfp;++m){
	 int n = m + f*p_Nfp + p_Nfp*p_Nfaces*k;
	 int idM = mesh->vmapM[n];
	 int idP = mesh->vmapP[n];
	 int  nM = idM%p_Np; 
	 int  nP = idP%p_Np; 
	 int  kM = (idM-nM)/p_Np;
	 int  kP = (idP-nP)/p_Np;
	 idM = nM + Nfields*BSIZE*kM;
	 idP = nP + Nfields*BSIZE*kP;
	 
	 /* stub resolve some other way */
	 if(mesh->vmapP[n]<0){
	   idP = mesh->vmapP[n]; /* -ve numbers */
	 }
 
	 sk = 6*p_Nfp*p_Nfaces*k+m+f*p_Nfp;
	 h_surfinfo[sk + 0*p_Nfp*p_Nfaces] = idM;
	 h_surfinfo[sk + 1*p_Nfp*p_Nfaces] = idP;
	 h_surfinfo[sk + 2*p_Nfp*p_Nfaces] = sJk[f]/(2.*J);
	 h_surfinfo[sk + 3*p_Nfp*p_Nfaces] = (idM==idP)?-1.:1.; 
	 h_surfinfo[sk + 4*p_Nfp*p_Nfaces] = nxk[f];
	 h_surfinfo[sk + 5*p_Nfp*p_Nfaces] = nyk[f];
       }
     }
  }
   
   hipMalloc  ((void**) &c_surfinfo, sz);
   hipMemcpy( c_surfinfo, h_surfinfo, sz, hipMemcpyHostToDevice);

   hipBindTexture(0,  t_surfinfo, c_surfinfo, sz);

   free(h_surfinfo);

   sz = mesh->parNtotalout*sizeof(int);
   hipMalloc((void**) &(mesh->c_parmapOUT), sz);
   hipMemcpy(mesh->c_parmapOUT,  mesh->parmapOUT, sz, hipMemcpyHostToDevice);

   return dt;
#endif
}



__global__ void MaxwellsGPU_VOL_Kernel2D(float *g_rhsQ, float2 *g_DrDs){

  /* fastest */
  __device__ __shared__ float s_Q[p_Nfields*BSIZE];
  __device__ __shared__ float s_facs[4];

  /* LOCKED IN to using Np threads per block */
  const int n = threadIdx.x;
  const int k = blockIdx.x;
  
  /* "coalesced"  */
  int m = n+k*p_Nfields*BSIZE;
  int id = n;
  s_Q[id] = tex1Dfetch(t_Q, m); m+=BSIZE; id+=BSIZE;
  s_Q[id] = tex1Dfetch(t_Q, m); m+=BSIZE; id+=BSIZE;
  s_Q[id] = tex1Dfetch(t_Q, m); 

#if 1
  if(p_Np<4 && n==0)
    for(m=0;m<4;++m)
      s_facs[m] = tex1Dfetch(t_vgeo, 4*k+m);
  else if((n<4) && (p_Np>=4))
    s_facs[n] = tex1Dfetch(t_vgeo, 4*k+n);
#else
  if(n==0)
    for(m=0;m<4;++m)
      s_facs[m] = tex1Dfetch(t_vgeo, 4*k+m);
#endif
  __syncthreads();

  float dHxdr=0,dHxds=0;
  float dHydr=0,dHyds=0;
  float dEzdr=0,dEzds=0;

  float Q;
  for(m=0;p_Np-m;){
    float4 D = tex1Dfetch(t_DrDs, n+m*BSIZE);

    id = m;
    Q = s_Q[id]; dHxdr += D.x*Q; dHxds += D.y*Q;  id += BSIZE;
    Q = s_Q[id]; dHydr += D.x*Q; dHyds += D.y*Q;  id += BSIZE;
    Q = s_Q[id]; dEzdr += D.x*Q; dEzds += D.y*Q;  
    ++m;

#if (p_Np%2) == 0
    id = m;
    Q = s_Q[id]; dHxdr += D.z*Q; dHxds += D.w*Q;  id += BSIZE;
    Q = s_Q[id]; dHydr += D.z*Q; dHyds += D.w*Q;  id += BSIZE;
    Q = s_Q[id]; dEzdr += D.z*Q; dEzds += D.w*Q;  
    ++m;
#endif
  }
  
  const float drdx= s_facs[0];
  const float drdy= s_facs[1];
  const float dsdx= s_facs[2];
  const float dsdy= s_facs[3];

  m = n+p_Nfields*BSIZE*k;
  if(n<BSIZE){
  g_rhsQ[m] = -(drdy*dEzdr+dsdy*dEzds); m += BSIZE;
  g_rhsQ[m] =  (drdx*dEzdr+dsdx*dEzds); m += BSIZE;
  g_rhsQ[m] =  (drdx*dHydr+dsdx*dHyds - drdy*dHxdr-dsdy*dHxds); 
  }
}

__global__ void MaxwellsGPU_SURF_Kernel2D(float *g_Q, float *g_rhsQ){

  __device__ __shared__ float s_fluxQ[p_Nfields*p_Nfp*p_Nfaces];

  /* LOCKED IN to using Np threads per block */
  const int n = threadIdx.x;
  const int k = blockIdx.x;
  int m;

  /* grab surface nodes and store flux in shared memory */
  if(n< (p_Nfp*p_Nfaces) ){
    /* coalesced reads (maybe) */
    m = 6*(k*p_Nfp*p_Nfaces)+n;
    const  int idM   = tex1Dfetch(t_surfinfo, m); m += p_Nfp*p_Nfaces;
           int idP   = tex1Dfetch(t_surfinfo, m); m += p_Nfp*p_Nfaces;
    const  float Fsc = tex1Dfetch(t_surfinfo, m); m += p_Nfp*p_Nfaces;
    const  float Bsc = tex1Dfetch(t_surfinfo, m); m += p_Nfp*p_Nfaces;
    const  float nx  = tex1Dfetch(t_surfinfo, m); m += p_Nfp*p_Nfaces;
    const  float ny  = tex1Dfetch(t_surfinfo, m); 

    /* check if idP<0  */
    float dHx=0, dHy=0, dEz=0;

    if(idP<0){
      idP = p_Nfields*(-1-idP);
      
      dHx = Fsc*(tex1Dfetch(t_partQ, idP+0) - tex1Dfetch(t_Q, idM+0*BSIZE));
      dHy = Fsc*(tex1Dfetch(t_partQ, idP+1) - tex1Dfetch(t_Q, idM+1*BSIZE));
      dEz = Fsc*(tex1Dfetch(t_partQ, idP+2) - tex1Dfetch(t_Q, idM+2*BSIZE));
    }
    else{
      dHx = Fsc*(    tex1Dfetch(t_Q, idP+0*BSIZE) - tex1Dfetch(t_Q, idM+0*BSIZE));
      dHy = Fsc*(    tex1Dfetch(t_Q, idP+1*BSIZE) - tex1Dfetch(t_Q, idM+1*BSIZE));
      dEz = Fsc*(Bsc*tex1Dfetch(t_Q, idP+2*BSIZE) - tex1Dfetch(t_Q, idM+2*BSIZE));
    }

    const float ndotdH = nx*dHx + ny*dHy;

    m = n;
    s_fluxQ[m] = -ny*dEz + dHx - ndotdH*nx; m += p_Nfp*p_Nfaces;
    s_fluxQ[m] =  nx*dEz + dHy - ndotdH*ny; m += p_Nfp*p_Nfaces;
    s_fluxQ[m] =  nx*dHy - ny*dHx + dEz;
  }

  /* make sure all element data points are cached */
  __syncthreads();

  if(n< (p_Np))
  {
    float rhsHx = 0, rhsHy = 0, rhsEz = 0;
    
    int sk = n;
    /* can manually unroll to 4 because there are 3 faces */
    for(m=0;p_Nfaces*p_Nfp-m;){
#if 0
      float4 L;
      L.x = tex1Dfetch(t_LIFT, n+    m*p_Np); 
      L.y = tex1Dfetch(t_LIFT, n+(m+1)*p_Np); 
      L.z = tex1Dfetch(t_LIFT, n+(m+2)*p_Np); 
#else
      float4 L = tex1Dfetch(t_LIFT, sk); sk+=p_Np;
#endif
      /* broadcast */
      int sk1 = m;
      rhsHx += L.x*s_fluxQ[sk1]; sk1 += p_Nfp*p_Nfaces;
      rhsHy += L.x*s_fluxQ[sk1]; sk1 += p_Nfp*p_Nfaces;
      rhsEz += L.x*s_fluxQ[sk1]; sk1 += p_Nfp*p_Nfaces;
      ++m;

      /* broadcast */
      sk1 = m;
      rhsHx += L.y*s_fluxQ[sk1]; sk1 += p_Nfp*p_Nfaces;
      rhsHy += L.y*s_fluxQ[sk1]; sk1 += p_Nfp*p_Nfaces;
      rhsEz += L.y*s_fluxQ[sk1]; sk1 += p_Nfp*p_Nfaces;
      ++m;

      /* broadcast */
      sk1 = m;
      rhsHx += L.z*s_fluxQ[sk1]; sk1 += p_Nfp*p_Nfaces;
      rhsHy += L.z*s_fluxQ[sk1]; sk1 += p_Nfp*p_Nfaces;
      rhsEz += L.z*s_fluxQ[sk1]; sk1 += p_Nfp*p_Nfaces;
      ++m;

    }
    
    m = n+k*p_Nfields*BSIZE;
    g_rhsQ[m] += rhsHx; m += BSIZE;
    g_rhsQ[m] += rhsHy; m += BSIZE;
    g_rhsQ[m] += rhsEz; 

  }
}


__global__ void MaxwellsGPU_RK_Kernel2D(int Ntotal, float *g_resQ, float *g_rhsQ, float *g_Q, float fa, float fb, float fdt){
  
  int n = blockIdx.x * blockDim.x + threadIdx.x;
    
  if(n<Ntotal){
    float rhs = g_rhsQ[n];
    float res = g_resQ[n];
    res = fa*res + fdt*rhs;
    
    g_resQ[n] = res;
    g_Q[n]    += fb*res;
  }

} 


/* assumes data resides on device */
void MaxwellsKernel2d(Mesh *mesh, float frka, float frkb, float fdt){

  /* grab data from device and initiate sends */
  MaxwellsMPISend2d(mesh);

  int ThreadsPerBlock, BlocksPerGrid;	

  BlocksPerGrid   = mesh->K; 
  ThreadsPerBlock = p_Np; 
  
  /* evaluate volume derivatives */
  MaxwellsGPU_VOL_Kernel2D <<< BlocksPerGrid, ThreadsPerBlock >>>  
    (c_rhsQ, c_DrDs2);

  /* finalize sends and recvs, and transfer to device */
  MaxwellsMPIRecv2d(mesh, c_partQ);

  BlocksPerGrid = mesh->K;

  if( ( p_Nfp*p_Nfaces ) > (p_Np) )
    ThreadsPerBlock = p_Nfp*p_Nfaces;
  else
    ThreadsPerBlock = p_Np;

  /* evaluate surface contributions */
  MaxwellsGPU_SURF_Kernel2D <<< BlocksPerGrid, ThreadsPerBlock >>>
    (c_Q, c_rhsQ);

  int Ntotal = mesh->K*BSIZE*p_Nfields;
  
  ThreadsPerBlock = 256;
  BlocksPerGrid = (Ntotal+ThreadsPerBlock-1)/ThreadsPerBlock;

  /* update RK Step */
  MaxwellsGPU_RK_Kernel2D<<< BlocksPerGrid, ThreadsPerBlock >>> 
    (Ntotal, c_resQ, c_rhsQ, c_Q, frka, frkb, fdt);

}




void gpu_set_data2d(int K,
		  double *d_Hx, double *d_Hy, double *d_Ez){


  float *f_Q = (float*) calloc(K*p_Nfields*BSIZE,sizeof(float));
  
  /* also load into usual data matrices */
  
  for(int k=0;k<K;++k){
    for(int n=0;n<p_Np;++n)
       f_Q[n       +k*BSIZE*p_Nfields] = d_Hx[n+k*p_Np];
    for(int n=0;n<p_Np;++n)
      f_Q[n  +BSIZE+k*BSIZE*p_Nfields] = d_Hy[n+k*p_Np];
    for(int n=0;n<p_Np;++n)
      f_Q[n+2*BSIZE+k*BSIZE*p_Nfields] = d_Ez[n+k*p_Np];
  }
  
  hipMemcpy(c_Q, f_Q, BSIZE*K*p_Nfields*sizeof(float), hipMemcpyHostToDevice);
  
  free(f_Q);
}
  
void gpu_get_data2d(int K,
		  double *d_Hx, double *d_Hy, double *d_Ez){

  float *f_Q = (float*) calloc(K*p_Nfields*BSIZE,sizeof(float));
  
  hipMemcpy(f_Q, c_Q, K*BSIZE*p_Nfields*sizeof(float), hipMemcpyDeviceToHost);

  /* also load into usual data matrices */
  
  for(int k=0;k<K;++k){
    for(int n=0;n<p_Np;++n)
      d_Hx[n+k*p_Np] = f_Q[n        +k*BSIZE*p_Nfields];
    for(int n=0;n<p_Np;++n) 
      d_Hy[n+k*p_Np] = f_Q[n  +BSIZE+k*BSIZE*p_Nfields];
    for(int n=0;n<p_Np;++n)
      d_Ez[n+k*p_Np] = f_Q[n+2*BSIZE+k*BSIZE*p_Nfields];

  }

  free(f_Q);
}

__global__ void partial_get_kernel(int Ntotal, int *g_index, float *g_partQ){
  
  int n = blockIdx.x * blockDim.x + threadIdx.x;
    
  if(n<Ntotal)
    g_partQ[n] = tex1Dfetch(t_Q, g_index[n]);
  
} 

void get_partial_gpu_data2d(int Ntotal, int *g_index, float *h_partQ){

  int ThreadsPerBlock = 256;
  int BlocksPerGrid = (Ntotal+ThreadsPerBlock-1)/ThreadsPerBlock;

  partial_get_kernel <<< BlocksPerGrid, ThreadsPerBlock >>> (Ntotal, g_index, c_tmp);

  hipMemcpy(h_partQ, c_tmp, Ntotal*sizeof(float), hipMemcpyDeviceToHost);
}


}
